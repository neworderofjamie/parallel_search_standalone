#include "hip/hip_runtime.h"
// Standard C++ includes
#include <algorithm>
#include <iostream>
#include <numeric>
#include <random>
#include <stdexcept>
#include <string>
#include <sstream>

// Standard C includes
#include <cassert>
#include <cmath>

// CUDA includes
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>

// Include nightmare array
#include "big_array.h"

//------------------------------------------------------------------------
// Macros
//------------------------------------------------------------------------
#define CHECK_CUDA_ERRORS(call) {                                                                   \
    hipError_t error = call;                                                                       \
    if (error != hipSuccess) {                                                                     \
            std::ostringstream errorMessageStream;                                                  \
            errorMessageStream << "cuda error:" __FILE__ << ": " << __LINE__ << " ";                \
            errorMessageStream << hipGetErrorString(error) << "(" << error << ")" << std::endl;    \
            throw std::runtime_error(errorMessageStream.str());                                     \
        }                                                                                           \
    }

template<typename T>
using HostDeviceArray = std::pair < T*, T* > ;

//-----------------------------------------------------------------------------
__global__ void testPerThreadBisect(unsigned int offset, uint16_t *d_outIndex, unsigned int *d_mergedPresynapticUpdateGroupStartID1)
{
    const unsigned int id = 32 * blockIdx.x + threadIdx.x; 
    
    const unsigned int offsetID = offset + id;
    
    unsigned int lo = 0;
    unsigned int hi = 62496;
    while(lo < hi)
    {
        const unsigned int mid = (lo + hi) / 2;
        if(offsetID < d_mergedPresynapticUpdateGroupStartID1[mid]) {
            hi = mid;
        }
        else {
            lo = mid + 1;
        }
    }
    
    // Write index to output array
    d_outIndex[id] = lo - 1;
}

//-----------------------------------------------------------------------------
// Host functions
//-----------------------------------------------------------------------------
template<typename T>
HostDeviceArray<T> allocateHostDevice(unsigned int count)
{
    T *array = nullptr;
    T *d_array = nullptr;
    CHECK_CUDA_ERRORS(hipHostMalloc(&array, count * sizeof(T)));
    CHECK_CUDA_ERRORS(hipMalloc(&d_array, count * sizeof(T)));

    return std::make_pair(array, d_array);
}
//-----------------------------------------------------------------------------
template<typename T>
void hostToDeviceCopy(HostDeviceArray<T> &array, unsigned int count, bool deleteHost=false)
{
    CHECK_CUDA_ERRORS(hipMemcpy(array.second, array.first, sizeof(T) * count, hipMemcpyHostToDevice));
    if (deleteHost) {
        CHECK_CUDA_ERRORS(hipHostFree(array.first));
        array.first = nullptr;
    }
}
//-----------------------------------------------------------------------------
template<typename T>
void deviceToHostCopy(HostDeviceArray<T> &array, unsigned int count)
{
    CHECK_CUDA_ERRORS(hipMemcpy(array.first, array.second, count * sizeof(T), hipMemcpyDeviceToHost));
}
//-----------------------------------------------------------------------------
int main()
{
    const unsigned int numGroups = sizeof(mergedPresynapticUpdateGroupStartID1) / sizeof(unsigned int);
    const unsigned int offset = 32923392;
    const unsigned int numThreads = 1017051648;
    CHECK_CUDA_ERRORS(hipSetDevice(0));

    // Create events
    hipEvent_t testStart;
    hipEvent_t testStop;
    CHECK_CUDA_ERRORS(hipEventCreate(&testStart));
    CHECK_CUDA_ERRORS(hipEventCreate(&testStop));

    // Create output array
    auto outIndex = allocateHostDevice<uint16_t>(numThreads);
    hostToDeviceCopy(outIndex, numThreads);

    // Create device version of presynaptic update group start ids
    unsigned int *d_mergedPresynapticUpdateGroupStartID1;
    CHECK_CUDA_ERRORS(hipMalloc(&d_mergedPresynapticUpdateGroupStartID1, numGroups * sizeof(unsigned int)));
    CHECK_CUDA_ERRORS(hipMemcpy(d_mergedPresynapticUpdateGroupStartID1, mergedPresynapticUpdateGroupStartID1, 
                                 sizeof(unsigned int) * numGroups, hipMemcpyHostToDevice));

    // Run kernel
    CHECK_CUDA_ERRORS(hipEventRecord(testStart));
    const dim3 threads(32, 1);
    const dim3 grid(31782864, 1);
    testPerThreadBisect<<<grid, threads>>>(offset, outIndex.second, d_mergedPresynapticUpdateGroupStartID1);
    CHECK_CUDA_ERRORS(hipPeekAtLastError());
    CHECK_CUDA_ERRORS(hipEventRecord(testStop));

    // Get kernel time
    float time;
    CHECK_CUDA_ERRORS(hipEventSynchronize(testStop));
    CHECK_CUDA_ERRORS(hipEventElapsedTime(&time, testStart, testStop));
    std::cout << "Search kernel takes " << time << "ms" << std::endl;

    // Copy output indices 
    deviceToHostCopy(outIndex, numThreads);

    // Verify binning
    unsigned int nextGroup = 1;
    for(unsigned int i = 0; i < numThreads; i++) {
        if((i + offset) == mergedPresynapticUpdateGroupStartID1[nextGroup]) {
            nextGroup++;
        }
        assert(outIndex.first[i] == nextGroup - 1);
    }
    std::cout << "Output correct!" << std::endl;

    return EXIT_SUCCESS;
}
